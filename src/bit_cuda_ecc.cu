#include "hip/hip_runtime.h"
#include "cuECC/ecc.cuh"
#include "cuECC/u256.cuh"
#include "cuECC/fp.cuh"
#include "cuECC/point.cuh"  // استفاده از تعریف موجود
#include "cuECC/secp256k1.cuh"
#include <iostream>

// حذف تعریف ساختار Point (چون توی point.cuh هست)

// جمع دو نقطه (مثال ساده)
__host__ __device__ Point operator+(const Point& p1, const Point& p2) {
    if (p1.infinity) return p2;
    if (p2.infinity) return p1;
    // اینجا باید پیاده‌سازی جمع نقاط رو از ecc.cu یا point.cuh بیاری
    return Point();
}

// کرنل CUDA برای ضرب نقطه‌ای
__global__ void multiply_point_kernel(Point* points, u256* scalars, Point* result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        Point p = points[idx];
        u256 scalar = scalars[idx];
        Point r = Point();
        for (int i = 255; i >= 0; i--) {
            r = r + r;
            if (scalar.get_bit(i)) {
                r = r + p;
            }
        }
        result[idx] = r;
    }
}

// تابع wrapper برای تولید کلید عمومی
Point generate_public_key(u256 private_key) {
    Point* d_points;
    u256* d_scalars;
    Point* d_result;
    Point result;

    hipMalloc(&d_points, sizeof(Point));
    hipMalloc(&d_scalars, sizeof(u256));
    hipMalloc(&d_result, sizeof(Point));

    Point G = secp256k1::generator; // فرض می‌کنیم این توی secp256k1.cuh تعریف شده

    hipMemcpy(d_points, &G, sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_scalars, &private_key, sizeof(u256), hipMemcpyHostToDevice);

    multiply_point_kernel<<<1, 1>>>(d_points, d_scalars, d_result, 1);

    hipMemcpy(&result, d_result, sizeof(Point), hipMemcpyDeviceToHost);

    hipFree(d_points);
    hipFree(d_scalars);
    hipFree(d_result);

    return result;
}

// تابع اصلی (برای تست)
extern "C" void run_ecc() {
    u256 private_key("0x1234567890abcdef1234567890abcdef1234567890abcdef1234567890abcdef");
    Point public_key = generate_public_key(private_key);
    std::cout << "Public Key X: " << public_key.x.to_string() << std::endl;
    std::cout << "Public Key Y: " << public_key.y.to_string() << std::endl;
}